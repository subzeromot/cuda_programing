#include "stdio.h"
#include<iostream>
#include <hip/hip_runtime.h>


#define N 50000 // number of element in array
// BUILD: nvcc -0 01_add_two_vector 01_add_two_vector.cu

__global__ void gpu_add(int *d_a, int* d_b, int *d_c){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N){
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void){
    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    /* cuda allocate the memmory*/
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    /* init input array */
    for(int i = 0; i < N; i++){
        h_a[i] = 2*i*i;
        h_b[i] = i;
    }
    // std::cout << "Vector 1: ";
    // for(int i = 0; i < N; i++){
    //     std::cout << h_a[i] << " ";
    // }
    // std::cout << std::endl;
    // std::cout << "Vector 2: ";
    // for(int i = 0; i < N; i++){
    //     std::cout << h_b[i] << " ";
    // }
    // std::cout << std::endl;

    /* copy array data from host to device */
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    /* call kernel function */
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    gpu_add<<<512,512>>> (d_a, d_b, d_c);
    hipDeviceSynchronize();
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    /* Copy data from device to host */
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // std::cout << "Output: ";
    // for(int i = 0; i < N; i++){
    //     std::cout << h_c[i] << " ";
    // }
    // std::cout << std::endl;

    int error = 0;
    for (int i = 0; i < N; i++)
    {
        if ((h_a[i] + h_b[i] != h_c[i]))
        { error++; }
    }
    std::cout << "Error: " << error << std::endl;

    std::cout << "GPU Times: " << elapsedTime << " ms" << std::endl;
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}